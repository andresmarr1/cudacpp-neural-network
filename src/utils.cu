//
// Created by andres on 2/4/25.
//
#include "utils.h"

void allocate_device_memory(float*& d_images, int*& d_labels, int batch_size){
  hipMalloc((void**)&d_images, sizeof(float)*batch_size*IMG_SIZE);
  hipMalloc((void**)&d_labels, sizeof(int)*batch_size);
}
void copy_batch_to_device(const std::vector<float>& batch_images,const std::vector<int>& batch_labels, const std::vector<float>& d_images, std::vector<ing>& d_labels){
  hipMemcpy(d_images, batch_images.data(), batch_images.size()*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_labels, batch_labels.data(), d_labels.size()*sizeof(int), hipMemcpyHostToDevice);
}
void free_device_memory(float* d_images, int* d_labels){
  hipFree(d_images);
  hipFree(d_labels);
}